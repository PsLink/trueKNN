#include "hip/hip_runtime.h"
#include "header.h"
// #include <float.h>


#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38
#endif


__device__ inline float euclid_dist_2(
    int clusterId,
    int dim,
    const short int* __restrict__ query,
    const float* __restrict__ clusters) 
{
    float ans = 0.0;

    for (int i = 0; i < dim; i++) {
        float diff = query[i] - clusters[i + clusterId * dim];
        ans += diff * diff;
    }

    return ans;
}

__global__ void calDistance (
    int dim, 
    int k, 
    const short int* __restrict__ query, 
    const float* __restrict__ clusters, 
    float* __restrict__ dis)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < k) {
        dis[tid] = euclid_dist_2(tid, dim, query, clusters);
    }
}


void dis(
    std::vector<short int> &query,
    std::vector<std::vector<short int> > &Centroids,
    std::vector<float> &distance,
    int range_start,
    int batch_size) 
{
    int dim = query.size(), k = batch_size;

    float *cluster;
    cluster = (float *)calloc(k * dim, sizeof(float));

    for (int i = 0; i < k; i++)
        for (int j = 0; j < dim; j++) {
            cluster[i * dim + j] = Centroids[i + range_start][j];
        }

    distance.resize(k);

    hipSetDevice(DEVICE_LOCAL_RANK);

    short int *d_query;
    hipMalloc((void**)&d_query, dim * sizeof(short int));
    hipMemcpy(d_query, query.data(), dim * sizeof(short int), hipMemcpyDefault);

    float *d_dis;
    hipMalloc((void**)&d_dis, k * sizeof(float));
    hipMemset(d_dis, 0, k * sizeof(float));

    float *d_clusters;
    hipMalloc((void**)&d_clusters, k * dim * sizeof(float));
    hipMemcpy(d_clusters, cluster, k * dim * sizeof(float), hipMemcpyDefault);

    int blockSize = 256;
    int gridSize = (k + blockSize - 1) / blockSize;
    // hipSetDevice(DEVICE_LOCAL_RANK);
    calDistance <<< gridSize, blockSize>>>(dim, k, d_query, d_clusters, d_dis);
    hipMemcpy(distance.data(), d_dis, k * sizeof(float), hipMemcpyDefault);

    hipFree(d_query);
    hipFree(d_dis);
    hipFree(d_clusters);
    
    free(cluster);
}